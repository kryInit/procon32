
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>

#define CHECK(call) {                                                      \
    const hipError_t error = call;                                        \
    if (error != hipSuccess) {                                            \
        printf("Error: %s%d, ", __FILE__, __LINE__);                       \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1);                                                           \
    }                                                                      \
}

using namespace std;

#define rep(i,n) for(int i=0; i < static_cast<int>(n); ++i)

template<class T = int>
struct Vec2 {
    T x, y;

    template<class S> inline Vec2& operator =(const Vec2<S>& other) {
        this->x = other.x;
        this->y = other.y;
        return *this;
    }
    template<class S> inline bool operator<(const Vec2<S>& another) const {
        return x == another.x ? y < another.y : x < another.x;
    }
    template<class S> inline bool operator==(const Vec2<S>& another) const {
        return x == another.x && y == another.y;
    }
    template<class S> inline bool operator!=(const Vec2<S>& another) const {
        return x != another.x || y != another.y;
    }
    template<class S> inline Vec2 operator+(const Vec2<S>& other) const {
        return { x + other.x, y + other.y };
    }
    template<class S> inline Vec2& operator +=(const Vec2<S>& other) {
        *this = *this+other;
        return *this;
    }
    template<class S> inline Vec2 operator -(const Vec2<S>& other) const {
        return { x - other.x, y - other.y };
    }
    template<class S> inline Vec2& operator -=(const Vec2<S>& other) {
        *this = *this - other;
        return *this;
    }
    template<class S> inline Vec2 operator *(S s) const {
        return { x*s, y*s };
    }
    template<class S> inline Vec2& operator *=(S s) {
        *this = *this*s;
        return *this;
    }
    template<class S> inline Vec2 operator /(S s) const {
        return { x/s, y/s };
    }
    template<class S> inline Vec2& operator /=(S s) {
        *this = *this/s;
        return *this;
    }


    Vec2() : x(0), y(0) {};
    template<class N, class M> Vec2(N _x, M _y) : x(static_cast<T>(_x)), y(static_cast<T>(_y)) {}

    void dump() const { cout << '(' << x << ", " << y << ')' << endl; }
};

template<class T>
std::ostream &operator<<(std::ostream &out, const Vec2<T> &v) {
    out << '(' << v.x << ", " << v.y << ')';
    return out;
}

template<class T = int>
struct Vec3 {
    union {
        struct { T x,y,z; };
        struct { T r,g,b; };
    };

    template<class S> inline Vec3& operator =(const Vec3<S>& other) {
        this->x = other.x;
        this->y = other.y;
        this->z = other.z;
        return *this;
    }
    template<class S> inline bool operator<(const Vec3<S>& another) const {
        return x == another.x ? (y == another.y ? z < another.z : y < another.y) : x < another.x;
    }
    template<class S> inline bool operator==(const Vec3<S>& another) const {
        return x == another.x && y == another.y && z == another.z;
    }
    template<class S> inline bool operator!=(const Vec3<S>& another) const {
        return x != another.x || y != another.y || z != another.z;
    }
    template<class S> inline Vec3 operator+(const Vec3<S>& other) const {
        return { x + other.x, y + other.y, z + other.z };
    }
    template<class S> inline Vec3& operator +=(const Vec3<S>& other) {
        *this = *this+other;
        return *this;
    }
    template<class S> inline Vec3 operator -(const Vec3<S>& other) const {
        return { x - other.x, y - other.y, z - other.z };
    }
    template<class S> inline Vec3& operator -=(const Vec3<S>& other) {
        *this = *this-other;
        return *this;
    }
    template<class S> inline Vec3 operator *(S s) const {
        return { x*s, y*s, z*s };
    }
    template<class S> inline Vec3& operator *=(S s) {
        *this = *this*s;
        return *this;
    }
    template<class S> inline Vec3 operator /(S s) const {
        return { x/s, y/s, z/s };
    }
    template<class S> inline Vec3& operator /=(S s) {
        *this = *this/s;
        return *this;
    }

    Vec3() : x(), y(), z() {}
    template<class N, class M, class L> Vec3(N _x, M _y, L _z) : x(static_cast<T>(_x)), y(static_cast<T>(_y)), z(static_cast<T>(_z)) {}

    void dump() { cout << '(' << x << ", " << y << ", " << z << ')' << endl; }
    double mag() {
        auto X = static_cast<double>(x);
        auto Y = static_cast<double>(y);
        auto Z = static_cast<double>(z);
        return sqrt(X*X+Y*Y+Z*Z);
    }
    template<class S>
    Vec3<T> mul_each_other(Vec3<S>& other) const {
        return { x*static_cast<T>(other.x), y*static_cast<T>(other.y), z*static_cast<T>(other.z) };
    }
    T sum() const { return x + y + z; }
};

template<class T>
std::ostream &operator<<(std::ostream &out, const Vec3<T> &v) {
    out << '(' << v.x << ", " << v.y << ", " << v.z << ')';
    return out;
}

using RGB = Vec3<unsigned char>;

template <>
void RGB::dump() {
    cout << static_cast<unsigned>(r) << ", " << static_cast<unsigned>(g) << ", " << static_cast<unsigned>(b) << endl;
}

ostream& operator <<(ostream& os, const RGB& rgb) {
    os << '(' << static_cast<unsigned>(rgb.r) << ", " << static_cast<unsigned>(rgb.g) << ", " << static_cast<unsigned>(rgb.b)<< ')';
    return os;
}

class ImageFragment {
    RGB *img_data;
public:
    const unsigned int FRAG_SIZE;

    // ImageFragment() : PIX_SIZE(0), img_data(nullptr) {}
    ImageFragment() = delete;
    explicit ImageFragment(unsigned int frag_size) : FRAG_SIZE(frag_size) {
        img_data = (RGB*)malloc(FRAG_SIZE*FRAG_SIZE*sizeof(RGB));
    }
    ~ImageFragment() {
        if (img_data != nullptr) free(img_data);
    }

    void input(const string& file_name) {
        ifstream ifs(file_name, ios::binary);
        ifs.read((char*)img_data, FRAG_SIZE*FRAG_SIZE*sizeof(RGB));
    }
    void dump() {
        rep(i,FRAG_SIZE) {
            cout << "[";
            rep(j,FRAG_SIZE) {
                cout << *(img_data+i*FRAG_SIZE+j) << ", ";
            }
            cout << "]" << endl;
        }
    }
    void pdump() {
        rep(i,FRAG_SIZE) {
            cout << "[";
            rep(j,FRAG_SIZE) {
                cout << (img_data+i*FRAG_SIZE+j) << ", ";
            }
            cout << "]" << endl;
        }

    }

    unsigned int area() const { return FRAG_SIZE*FRAG_SIZE; }

    void copy_img_data_for_cuda(RGB* img_data_cuda) const {
        hipMemcpy(img_data_cuda, img_data, FRAG_SIZE*FRAG_SIZE*sizeof(RGB), hipMemcpyHostToDevice);
    }
    void simple_ave_conv(const unsigned int HALF_FILTER_SIZE, const unsigned int W_FOR_VAR, Vec3<float> *result) const {
        // result[4][FRAG_SIZE][W_FOR_VAR]
        rep(k,4) rep(i,FRAG_SIZE) rep(j,W_FOR_VAR) {
            int cy,cx;
            if (k == 0) cy = j, cx = i;
            else if (k == 1) cy = i, cx = FRAG_SIZE-j-1;
            else if (k == 2) cy = FRAG_SIZE-j-1, cx = FRAG_SIZE-i-1;
            else  cy = FRAG_SIZE-i-1, cx = j;

            Vec3<double> sum;
            int ly = max(0, cy-(int)HALF_FILTER_SIZE), ry = min(FRAG_SIZE-1, cy+HALF_FILTER_SIZE);
            int lx = max(0, cx-(int)HALF_FILTER_SIZE), rx = min(FRAG_SIZE-1, cx+HALF_FILTER_SIZE);
            for (int y = ly; y<=ry; y++) {
                for (int x = lx; x<=rx; ++x) {
                    sum += *(img_data + y*FRAG_SIZE + x);
                }
            }
            *(result + k*FRAG_SIZE*W_FOR_VAR + i*W_FOR_VAR + j) = sum / ((ry-ly+1)*(rx-lx+1));
        }
    }
};

class Image {
    ImageFragment *img_frags;

public:
    const Vec2<> DIV_NUM;
    const unsigned int FRAG_SIZE;

    Image(unsigned int w_div, unsigned int y_div, unsigned int frag_size, const string& frags_dir_path) : Image(Vec2<>(w_div, y_div), frag_size, frags_dir_path) {}
    Image(Vec2<> div_num, unsigned int frag_size, const string& frags_dir_path) : DIV_NUM(div_num), FRAG_SIZE(frag_size) {
        img_frags = (ImageFragment*)malloc(DIV_NUM.x*DIV_NUM.y*sizeof(ImageFragment));
        for (int i=0; i<DIV_NUM.y; ++i) {
            for (int j=0; j<DIV_NUM.x; ++j) {
                new(img_frags+i*DIV_NUM.x+j) ImageFragment(frag_size);
            }
        }

        for (int y=0; y<DIV_NUM.y; ++y) {
            for (int x=0; x<DIV_NUM.x; ++x) {
                stringstream ss;
                ss << frags_dir_path << hex << uppercase << x << y << ".ppm.bin";
                (img_frags+y*DIV_NUM.x+x)->input(ss.str());
            }
        }
    }

    unsigned int frag_num() const { return DIV_NUM.y * DIV_NUM.x; }

    void dump() {
        for(int i=0; i<DIV_NUM.y; ++i) {
            for (int j=0; j<DIV_NUM.x; ++j) {
                cout << "img_flags[" << i << "][" << j << "]: " << endl;
                (img_frags+i*DIV_NUM.x+j)->dump();
                cout << endl;
            }
        }
    }
    void copy_img_data_for_cuda(RGB* img_data_cuda) const {
        for (int idx=0; idx<DIV_NUM.y*DIV_NUM.x; ++idx) {
            (img_frags + idx)->copy_img_data_for_cuda(img_data_cuda + idx*((img_frags+idx)->area()));
        }
    }
    void simple_ave_conv(const unsigned int HALF_FILTER_SIZE, const unsigned int W_FOR_VAR, Vec3<float> *result) const {
        // result[4][FRAG_SIZE][W_FOR_VAR]
        // result[DIV_NUM.y][DIV_NUM.x][4][FRAG_SIZE][W_FOR_VAR]
        printf("result[%u][%u][4][%u][%u]\n", DIV_NUM.y, DIV_NUM.x, FRAG_SIZE, W_FOR_VAR);
        int one_result_size = 4*FRAG_SIZE*W_FOR_VAR;
        rep(i,DIV_NUM.y) rep(j,DIV_NUM.x) {
            int idx = i*DIV_NUM.x + j;
            cout << idx << endl;
            (img_frags + idx)->simple_ave_conv(HALF_FILTER_SIZE, W_FOR_VAR, result + idx*one_result_size);
        }
    }

    ~Image() {
        if (img_frags != nullptr) free(img_frags);
    }
};

class AdjacencyEvaluator {
public:
    const unsigned int HALF_FILTER_SIZE, W_FOR_VAR;

    AdjacencyEvaluator() : HALF_FILTER_SIZE(4), W_FOR_VAR(2) {}
    AdjacencyEvaluator(unsigned int half_filter_size, unsigned int w_for_var) : HALF_FILTER_SIZE(half_filter_size), W_FOR_VAR(w_for_var) {}

    void calc_similarity_simply(const Image &img) {
        const unsigned int ave_conv_result_size = img.frag_num()*4*img.FRAG_SIZE*W_FOR_VAR;
        auto *ave_conv_result = (Vec3<float>*)malloc(ave_conv_result_size*sizeof(Vec3<float>));
        img.simple_ave_conv(HALF_FILTER_SIZE, W_FOR_VAR, ave_conv_result);

        rep(i,img.DIV_NUM.y) rep(j,img.DIV_NUM.x) {
            cout << *(ave_conv_result + (i*img.DIV_NUM.x + j)*4*img.FRAG_SIZE*W_FOR_VAR) << endl;
        }

        const unsigned int similarity_size = img.frag_num()*4*img.frag_num()*4;
        auto similarity = (double*)malloc(similarity_size*sizeof(double));
        auto sum_memo = (Vec3<double>*)malloc(img.FRAG_SIZE*sizeof(Vec3<double>));
        auto s_sum_memo = (Vec3<double>*)malloc(img.FRAG_SIZE*sizeof(Vec3<double>));
        double *now_ptr = similarity;
        rep(i,img.DIV_NUM.y) rep(j,img.DIV_NUM.x) rep(k,4) {
            {
                unsigned int offset = (i*img.DIV_NUM.x*4+j*4+k)*img.FRAG_SIZE*W_FOR_VAR;
                rep(y, img.FRAG_SIZE) {
                    sum_memo[img.FRAG_SIZE-y-1] = {0,0,0};
                    s_sum_memo[img.FRAG_SIZE-y-1] = {0,0,0};
                    rep(x, W_FOR_VAR) {
                        Vec3<double> tmp;
                        tmp += *(ave_conv_result + offset + y*W_FOR_VAR+x);
                        sum_memo[img.FRAG_SIZE-y-1] += tmp;
                        s_sum_memo[img.FRAG_SIZE-y-1] += tmp.mul_each_other(tmp);
                    }
                }
            }
            rep(I,img.DIV_NUM.y) rep(J,img.DIV_NUM.x) {
                if (i == I && j == J) {
                    rep(K,4) {
                        *now_ptr = INFINITY;
                        now_ptr++;
                    }
                    continue;
                }
                rep(K,4) {
                    unsigned int offset = (I*img.DIV_NUM.x*4+J*4+K)*img.FRAG_SIZE*W_FOR_VAR;
                    double var_sum = 0;
                    rep(y, img.FRAG_SIZE){
                        Vec3<double> sum = sum_memo[y];
                        Vec3<double> s_sum = s_sum_memo[y];
                        rep(x, W_FOR_VAR) {
                            Vec3<double> tmp;
                            tmp += *(ave_conv_result + offset + y*W_FOR_VAR+x);
                            sum += tmp;
                            s_sum += tmp.mul_each_other(tmp);
                        }
                        double tmp = ((s_sum - (sum.mul_each_other(sum))/(2*W_FOR_VAR)) / (2*W_FOR_VAR)).sum() / 3.;
                        var_sum += tmp;
                    }
                    double ave_var = var_sum / img.FRAG_SIZE;
                    *now_ptr = ave_var;
                    now_ptr++;
                }
            }
        }
        rep(i,img.DIV_NUM.y) {
            rep(j,img.DIV_NUM.x) {
                rep(k,4) {
                    const unsigned int offset = (i*img.DIV_NUM.x*4+j*4+k)*img.frag_num()*4;
                    double best_similarity = 1e10;
                    int best_i, best_j, best_k;
                    rep(I,img.DIV_NUM.y) {
                        rep(J,img.DIV_NUM.x) {
                            if (I == i && J == j) continue;
                            rep(K,4) {
                                double tmp = *(similarity+offset + I*img.DIV_NUM.x*4 + J*4 + K);
                                if (best_similarity > tmp) {
                                    best_similarity = tmp;
                                    best_i = I;
                                    best_j = J;
                                    best_k = K;
                                }
                            }
                        }
                    }
                    cout << "(" << i << ", " << j << ", " << k <<  ") <=> (" << best_i << ", " << best_j << ", " << best_k << ") : " << best_similarity << endl;
                }
            }
        }
//        const unsigned int offset = (img.DIV_NUM.y*4 + 4);
        const unsigned int offset = 0;
        rep(i,img.DIV_NUM.y) {
            rep(j,img.DIV_NUM.x) {
                cout << "(";
                rep(k,4) cout << setw(2) << *(similarity + offset + i*img.DIV_NUM.x*4 + j*4+k) << ", ";
                cout << "), ";
            }
            cout << endl;
        }
        cout << endl;
    }
};

__global__ void kr(RGB *rgbs, int a);

int main(int argc, char *argv[]) {
#pragma omp parallel
    {
        std::cout << "Hello World!\n";
    }
    string frag_dir_path = string(argv[1]) + "/frags/";

    unsigned int h_div, w_div, frag_size;
    {
        unsigned int h_pix, w_pix, _;
        ifstream ifs("../prob.txt");
        ifs >> w_div >> h_div;
        ifs >> _ >> _ >> _;
        ifs >> w_pix >> h_pix;
        if (h_pix * w_div != w_pix * h_div) {
            cerr << "h_pix / h_div != w_pix / w_div" << endl;
            return -1;
        }
        frag_size = h_pix / h_div;
    }
    Image img(w_div, h_div, frag_size, frag_dir_path);
    img.dump();

    RGB *rgbs_cuda;
    hipMalloc(&rgbs_cuda, h_div*w_div*frag_size*frag_size*sizeof(RGB));
    img.copy_img_data_for_cuda(rgbs_cuda);
    kr<<<1,1>>>(rgbs_cuda, h_div*w_div*frag_size*frag_size-1);
    hipFree(rgbs_cuda);

    AdjacencyEvaluator ae(4, 4);
    ae.calc_similarity_simply(img);
}

__global__ void kr(RGB *rgbs, int a) {
    printf("%u %u %u\n", rgbs->r, rgbs->g, rgbs->b);
    rgbs += a;
    printf("%u %u %u\n", rgbs->r, rgbs->g, rgbs->b);
}
