#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <vec.hpp>
#include <time_manager.hpp>

#define CHECK(call) {                                                      \
    const hipError_t error = call;                                        \
    if (error != hipSuccess) {                                            \
        printf("Error: %s%d, ", __FILE__, __LINE__);                       \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1);                                                           \
    }                                                                      \
}

using namespace std;

#define rep(i,n) for(int i=0; i < static_cast<int>(n); ++i)

class ImageFragment {
    RGB *img_data;
public:
    const unsigned int FRAG_SIZE;

    // ImageFragment() : PIX_SIZE(0), img_data(nullptr) {}
    ImageFragment() = delete;
    explicit ImageFragment(unsigned int frag_size) : FRAG_SIZE(frag_size) {
        img_data = (RGB*)malloc(FRAG_SIZE*FRAG_SIZE*sizeof(RGB));
    }
    ~ImageFragment() {
        if (img_data != nullptr) free(img_data);
    }

    void input(const string& file_name) {
        ifstream ifs(file_name, ios::binary);
        ifs.read((char*)img_data, FRAG_SIZE*FRAG_SIZE*sizeof(RGB));
    }
    void dump() {
        rep(i,FRAG_SIZE) {
            cout << "[";
            rep(j,FRAG_SIZE) {
                cout << *(img_data+i*FRAG_SIZE+j) << ", ";
            }
            cout << "]" << endl;
        }
    }
    void pdump() {
        rep(i,FRAG_SIZE) {
            cout << "[";
            rep(j,FRAG_SIZE) {
                cout << (img_data+i*FRAG_SIZE+j) << ", ";
            }
            cout << "]" << endl;
        }

    }

    unsigned int area() const { return FRAG_SIZE*FRAG_SIZE; }

    void copy_img_data_for_cuda(RGB* img_data_cuda) const {
        hipMemcpy(img_data_cuda, img_data, FRAG_SIZE*FRAG_SIZE*sizeof(RGB), hipMemcpyHostToDevice);
    }
    void simple_ave_conv(const unsigned int HALF_FILTER_SIZE, const unsigned int W_FOR_VAR, Vec3<float> *result) const {
        // result[4][FRAG_SIZE][W_FOR_VAR]
        rep(k,4) rep(i,FRAG_SIZE) rep(j,W_FOR_VAR) {
            int cy,cx;
            if (k == 0) cy = j, cx = i;
            else if (k == 1) cy = i, cx = FRAG_SIZE-j-1;
            else if (k == 2) cy = FRAG_SIZE-j-1, cx = FRAG_SIZE-i-1;
            else  cy = FRAG_SIZE-i-1, cx = j;

            Vec3<double> sum;
            int ly = max(0, cy-(int)HALF_FILTER_SIZE), ry = min(FRAG_SIZE-1, cy+HALF_FILTER_SIZE);
            int lx = max(0, cx-(int)HALF_FILTER_SIZE), rx = min(FRAG_SIZE-1, cx+HALF_FILTER_SIZE);
            for (int y = ly; y<=ry; y++) {
                for (int x = lx; x<=rx; ++x) {
                    sum += *(img_data + y*FRAG_SIZE + x);
                }
            }
            *(result + k*FRAG_SIZE*W_FOR_VAR + i*W_FOR_VAR + j) = sum / ((ry-ly+1)*(rx-lx+1));
        }
    }
};

class Image {
    ImageFragment *img_frags;

public:
    const Vec2<> DIV_NUM;
    const unsigned int FRAG_SIZE;

    Image(unsigned int w_div, unsigned int y_div, unsigned int frag_size, const string& frags_dir_path) : Image(Vec2<>(w_div, y_div), frag_size, frags_dir_path) {}
    Image(Vec2<> div_num, unsigned int frag_size, const string& frags_dir_path) : DIV_NUM(div_num), FRAG_SIZE(frag_size) {
        img_frags = (ImageFragment*)malloc(DIV_NUM.x*DIV_NUM.y*sizeof(ImageFragment));
        for (int i=0; i<DIV_NUM.y; ++i) {
            for (int j=0; j<DIV_NUM.x; ++j) {
                new(img_frags+i*DIV_NUM.x+j) ImageFragment(frag_size);
            }
        }

        for (int y=0; y<DIV_NUM.y; ++y) {
            for (int x=0; x<DIV_NUM.x; ++x) {
                stringstream ss;
                ss << frags_dir_path << hex << uppercase << x << y << ".ppm.bin";
                (img_frags+y*DIV_NUM.x+x)->input(ss.str());
            }
        }
    }

    unsigned int frag_num() const { return DIV_NUM.y * DIV_NUM.x; }

    void dump() {
        for(int i=0; i<DIV_NUM.y; ++i) {
            for (int j=0; j<DIV_NUM.x; ++j) {
                cout << "img_flags[" << i << "][" << j << "]: " << endl;
                (img_frags+i*DIV_NUM.x+j)->dump();
                cout << endl;
            }
        }
    }
    void copy_img_data_for_cuda(RGB* img_data_cuda) const {
        for (int idx=0; idx<DIV_NUM.y*DIV_NUM.x; ++idx) {
            (img_frags + idx)->copy_img_data_for_cuda(img_data_cuda + idx*((img_frags+idx)->area()));
        }
    }
    void simple_ave_conv(const unsigned int HALF_FILTER_SIZE, const unsigned int W_FOR_VAR, Vec3<float> *result) const {
        // result[4][FRAG_SIZE][W_FOR_VAR]
        // result[DIV_NUM.y][DIV_NUM.x][4][FRAG_SIZE][W_FOR_VAR]
        printf("result[%u][%u][4][%u][%u]\n", DIV_NUM.y, DIV_NUM.x, FRAG_SIZE, W_FOR_VAR);
        int one_result_size = 4*FRAG_SIZE*W_FOR_VAR;
        rep(i,DIV_NUM.y) rep(j,DIV_NUM.x) {
            int idx = i*DIV_NUM.x + j;
            cout << idx << endl;
            (img_frags + idx)->simple_ave_conv(HALF_FILTER_SIZE, W_FOR_VAR, result + idx*one_result_size);
        }
    }

    ~Image() {
        if (img_frags != nullptr) free(img_frags);
    }
};

class AdjacencyEvaluator {
public:
    const unsigned int HALF_FILTER_SIZE, W_FOR_VAR;

    AdjacencyEvaluator() : HALF_FILTER_SIZE(4), W_FOR_VAR(2) {}
    AdjacencyEvaluator(unsigned int half_filter_size, unsigned int w_for_var) : HALF_FILTER_SIZE(half_filter_size), W_FOR_VAR(w_for_var) {}

    void calc_similarity_simply(const Image &img, double* similarity) {
        const unsigned int ave_conv_result_size = img.frag_num()*4*img.FRAG_SIZE*W_FOR_VAR;
        auto ave_conv_result = (Vec3<float>*)malloc(ave_conv_result_size*sizeof(Vec3<float>));
        img.simple_ave_conv(HALF_FILTER_SIZE, W_FOR_VAR, ave_conv_result);

        rep(i,img.DIV_NUM.y) rep(j,img.DIV_NUM.x) {
            cout << *(ave_conv_result + (i*img.DIV_NUM.x + j)*4*img.FRAG_SIZE*W_FOR_VAR) << endl;
        }

        // Σ(x_i - x_ave)^2 = ( Σ(x_i^2) - ((Σx_i)^2)/N ) / N
        auto sum_memo = (Vec3<double>*)malloc(img.FRAG_SIZE*sizeof(Vec3<double>));
        auto s_sum_memo = (Vec3<double>*)malloc(img.FRAG_SIZE*sizeof(Vec3<double>));
        double *now_ptr = similarity;
        rep(i,img.DIV_NUM.y) rep(j,img.DIV_NUM.x) rep(k,4) {
            {
                unsigned int offset = (i*img.DIV_NUM.x*4+j*4+k)*img.FRAG_SIZE*W_FOR_VAR;
                rep(y, img.FRAG_SIZE) {
                    sum_memo[img.FRAG_SIZE-y-1] = {0,0,0};
                    s_sum_memo[img.FRAG_SIZE-y-1] = {0,0,0};
                    rep(x, W_FOR_VAR) {
                        Vec3<double> tmp;
                        tmp += *(ave_conv_result + offset + y*W_FOR_VAR+x);
                        sum_memo[img.FRAG_SIZE-y-1] += tmp;
                        s_sum_memo[img.FRAG_SIZE-y-1] += tmp.mul_each_other(tmp);
                    }
                }
            }
            rep(I,img.DIV_NUM.y) rep(J,img.DIV_NUM.x) {
                if (i == I && j == J) {
                    rep(K,4) {
                        *now_ptr = INFINITY;
                        now_ptr++;
                    }
                    continue;
                }
                rep(K,4) {
                    unsigned int offset = (I*img.DIV_NUM.x*4+J*4+K)*img.FRAG_SIZE*W_FOR_VAR;
                    double var_sum = 0;
                    rep(y, img.FRAG_SIZE){
                        Vec3<double> sum = sum_memo[y];
                        Vec3<double> s_sum = s_sum_memo[y];
                        rep(x, W_FOR_VAR) {
                            Vec3<double> tmp;
                            tmp += *(ave_conv_result + offset + y*W_FOR_VAR+x);
                            sum += tmp;
                            s_sum += tmp.mul_each_other(tmp);
                        }
                        double tmp = ((s_sum - (sum.mul_each_other(sum))/(2*W_FOR_VAR)) / (2*W_FOR_VAR)).sum() / 3.;
                        var_sum += tmp;
                    }
                    double ave_var = var_sum / img.FRAG_SIZE;
                    *now_ptr = ave_var;
                    now_ptr++;
                }
            }
        }
        rep(i,img.DIV_NUM.y) {
            rep(j,img.DIV_NUM.x) {
                rep(k,4) {
                    const unsigned int offset = (i*img.DIV_NUM.x*4+j*4+k)*img.frag_num()*4;
                    double best_similarity = 1e10;
                    int best_i, best_j, best_k;
                    rep(I,img.DIV_NUM.y) {
                        rep(J,img.DIV_NUM.x) {
                            if (I == i && J == j) continue;
                            rep(K,4) {
                                double tmp = *(similarity+offset + I*img.DIV_NUM.x*4 + J*4 + K);
                                if (best_similarity > tmp) {
                                    best_similarity = tmp;
                                    best_i = I;
                                    best_j = J;
                                    best_k = K;
                                }
                            }
                        }
                    }
                    cout << "(" << i << ", " << j << ", " << k <<  ") <=> (" << best_i << ", " << best_j << ", " << best_k << ") : " << best_similarity << endl;
                }
            }
        }
        const unsigned int offset = (2*img.DIV_NUM.x*4 + 1*4 + 3)*img.frag_num()*4;
//        const unsigned int offset = 0;
        rep(i,img.DIV_NUM.y) {
            rep(j,img.DIV_NUM.x) {
                cout << "(";
                rep(k,4) cout << setw(2) << *(similarity + offset + i*img.DIV_NUM.x*4 + j*4+k) << ", ";
                cout << "), ";
            }
            cout << endl;
        }
        cout << endl;

        free(ave_conv_result);
        free(sum_memo);
        free(s_sum_memo);
    }
};

__global__ void kr(RGB *rgbs, int a);

int main(int argc, char *argv[]) {
#pragma omp parallel
    {
        std::cout << "Hello World!\n";
    }
    string frag_dir_path = string(argv[1]) + "/frags/";

    unsigned int h_div, w_div, frag_size;
    {
        unsigned int h_pix, w_pix, _;
        ifstream ifs(string(argv[1]) + "/prob.txt");
        ifs >> w_div >> h_div;
        ifs >> _ >> _ >> _;
        ifs >> w_pix >> h_pix;
        if (h_pix * w_div != w_pix * h_div) {
            cerr << "h_pix / h_div != w_pix / w_div" << endl;
            return -1;
        }
        frag_size = h_pix / h_div;
    }
    Image img(w_div, h_div, frag_size, frag_dir_path);
    img.dump();

    RGB *rgbs_cuda;
    hipMalloc(&rgbs_cuda, h_div*w_div*frag_size*frag_size*sizeof(RGB));
    img.copy_img_data_for_cuda(rgbs_cuda);
    kr<<<1,1>>>(rgbs_cuda, h_div*w_div*frag_size*frag_size-1);
    hipFree(rgbs_cuda);

    const unsigned int similarity_size = img.frag_num()*4*img.frag_num()*4;
    auto similarity = (double*)malloc(similarity_size*sizeof(double));
    AdjacencyEvaluator ae(2, 2);
    ae.calc_similarity_simply(img, similarity);

    free(similarity);
}

__global__ void kr(RGB *rgbs, int a) {
    printf("%u %u %u\n", rgbs->r, rgbs->g, rgbs->b);
    rgbs += a;
    printf("%u %u %u\n", rgbs->r, rgbs->g, rgbs->b);
}
